#include "hip/hip_runtime.h"
/*
Adapted from fractal code for CS 4380 / CS 5351

Copyright (c) 2018, Texas State University. All rights reserved.

Redistribution and usage in source and binary form, with or without
modification, is only permitted for educational use.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR
ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
(INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON
ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
(INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

Author: Martin Burtscher
Revision history:
20190610   andreainfufsm   Replaced the function to calculate the color of each
pixel
*/

#include <math.h>
#include <sys/time.h>

#include <cstdlib>

#include "wave.h"

__global__ void calculateframe(int frames, int width, unsigned char* pic) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  int N = frames * width * width;

  for (int i = index; i < N; i += stride) {
    int frame = i / (width * width);
    int idxInFrame = i % (width * width);
    int row = idxInFrame % width;
    int col = idxInFrame / width;

    float fx = col - 1024 / 2;
    float fy = row - 1024 / 2;
    float d = sqrtf(fx * fx + fy * fy);
    unsigned char color =
        (unsigned char)(160.0f + 127.0f * cos(d / 10.0f - frame / 7.0f) /
                                     (d / 50.0f + 1.0f));

    pic[i] = (unsigned char)color;
  }
}

int main(int argc, char* argv[]) {
  // check command line
  if (argc != 3) {
    fprintf(stderr, "usage: %s frame_width num_frames\n", argv[0]);
    exit(-1);
  }
  int width = atoi(argv[1]);
  if (width < 100) {
    fprintf(stderr, "error: frame_width must be at least 100\n");
    exit(-1);
  }
  int frames = atoi(argv[2]);
  if (frames < 1) {
    fprintf(stderr, "error: num_frames must be at least 1\n");
    exit(-1);
  }
  printf("computing %d frames of %d by %d picture\n", frames, width, width);

  int N = frames * width * width;
  unsigned char* pic;
  hipMallocManaged(&pic, N);

  // start time
  timeval start, end;
  gettimeofday(&start, NULL);

  int blockSize = 256;
  int numBlocks = (N + blockSize - 1) / blockSize;
  calculateframe<<<numBlocks, blockSize>>>(frames, width, pic);
  hipDeviceSynchronize();

  // end time
  gettimeofday(&end, NULL);
  double runtime = end.tv_sec + end.tv_usec / 1000000.0 - start.tv_sec -
                   start.tv_usec / 1000000.0;
  printf("compute time: %.4f s\n", runtime);

  // verify result by writing frames to BMP files
  if ((width <= 256) && (frames <= 100)) {
    for (int frame = 0; frame < frames; frame++) {
      char name[32];
      sprintf(name, "wave%d.bmp", frame + 1000);
      writeBMP(width, width, &pic[frame * width * width], name);
    }
  }

  hipFree(pic);
  return 0;
}